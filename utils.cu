#include "hip/hip_runtime.h"
#include "utils.h"

using namespace std;
std::vector<hipfftComplex> PCcoef(double BandWidth, double PulseWidth, double Fs, int _NFFT) {
//    std::cout << "BandWidth:" << BandWidth << " PulseWidth:" << PulseWidth << " Fs:" << Fs << std::endl;
    double Ts = 1 / Fs;

    int N = round(PulseWidth * Fs);
    double dT = (PulseWidth - Ts) / (N - 1); // t = linspace(-PulseWidth/2, PulseWidth/2-Ts, N);

    std::vector<hipfftComplex> result(_NFFT, make_hipComplex(0, 0));
    auto window = hammingWindow(N);
    // 生成线性调频信号
    for (int i = 0; i < N; ++i) {
        double t = -PulseWidth / 2 + i * dT;
        double phase = M_PI * BandWidth / PulseWidth * t * t;
        result[N-1-i] = hipConjf(make_hipComplex(cos(phase), sin(phase)));
    }

    for (int i = 0; i < N; ++i) {
        result[i].x = result[i].x * window[i];
        result[i].y = result[i].y * window[i];
    }

    return result;
}


unsigned int nextpow2(unsigned int x) {
    if (x == 0) return 1;
    return 1 << static_cast<unsigned int>(std::ceil(std::log2(x)));
}

bool isCudaAvailable() {
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        std::cout << "hipGetDeviceCount returned " << static_cast<int>(error_id) << "\n-> " << hipGetErrorString(error_id) << std::endl;
        return false;
    }

    if (deviceCount == 0) {
        std::cout << "There are no available device(s) that support CUDA" << std::endl;
        return false;
    } else {
        std::cout << "Detected " << deviceCount << " CUDA Capable device(s)" << std::endl;
        return true;
    }
}

long long calculateDuration(std::chrono::high_resolution_clock::time_point start, std::chrono::high_resolution_clock::time_point end) {
    // return duration time (us)
    using namespace std::chrono;
    auto duration = duration_cast<microseconds>(end - start);
    return duration.count();
}


void checkCufftErrors(hipfftResult result) {
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: ";
        switch (result) {
            case HIPFFT_INVALID_PLAN:
                std::cerr << "HIPFFT_INVALID_PLAN";
                break;
            case HIPFFT_ALLOC_FAILED:
                std::cerr << "HIPFFT_ALLOC_FAILED";
                break;
            case HIPFFT_INVALID_TYPE:
                std::cerr << "HIPFFT_INVALID_TYPE";
                break;
            case HIPFFT_INVALID_VALUE:
                std::cerr << "HIPFFT_INVALID_VALUE";
                break;
            case HIPFFT_INTERNAL_ERROR:
                std::cerr << "HIPFFT_INTERNAL_ERROR";
                break;
            case HIPFFT_EXEC_FAILED:
                std::cerr << "HIPFFT_EXEC_FAILED";
                break;
            case HIPFFT_SETUP_FAILED:
                std::cerr << "HIPFFT_SETUP_FAILED";
                break;
            case HIPFFT_INVALID_SIZE:
                std::cerr << "HIPFFT_INVALID_SIZE";
                break;
            default:
                std::cerr << "Unknown error";
        }
        std::cerr << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCudaErrors(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "hipError_t code: " << result << std::endl;
        throw std::runtime_error(hipGetErrorString(result));
    }
}

std::vector<double> hammingWindow(int N) {
    std::vector<double> window(N);
    for(int n = 0; n < N; n++) {
        window[n] = 0.54 - 0.46 * std::cos(2.0 * M_PI * n / (N - 1));
    }
    return window;
}

bool isEqual(double a, double b, double epsilon) {
    return fabs(a - b) < epsilon;
}

void getCurrentTime(uint32_t& second, uint32_t& microsecond) {
    struct timeval tv;
    gettimeofday(&tv, nullptr);
    second = static_cast<uint32_t>(tv.tv_sec % 86400);
    microsecond = static_cast<uint32_t>(tv.tv_usec);
}

unsigned int FourChars2Uint(char *startAddr) {
    return static_cast<uint8_t>(startAddr[0]) << 24
           | static_cast<uint8_t>(startAddr[1]) << 16
           | static_cast<uint8_t>(startAddr[2]) << 8
           | static_cast<uint8_t>(startAddr[3]);
}

void saveToBinaryFile(const hipfftComplex* d_data, size_t size, const char* filename) {
    auto* h_data = new hipfftComplex[size]; // 在主机上分配内存

    // 将数据从显存复制到主机内存
    hipMemcpy(h_data, d_data, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // 打开文件并以二进制方式写入
    std::ofstream file(filename, std::ios::binary);
    if (file.is_open()) {
        file.write(reinterpret_cast<char*>(h_data), size * sizeof(hipfftComplex));
        file.close();
        std::cout << "Data saved to " << filename << std::endl;
    } else {
        std::cerr << "Failed to open file for writing." << std::endl;
    }

    delete[] h_data; // 释放主机内存
}
