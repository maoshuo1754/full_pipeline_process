#include "hip/hip_runtime.h"
#include "WaveGroupProcessor.h"
#include "utils.h"
#include "SharedQueue.h"
#include <vector>

hipfftHandle WaveGroupProcessor::pc_plan_ = 0;            // 脉压FFT，用于对下面两个系数做脉压
hipfftComplex* WaveGroupProcessor::d_pc_coeffs_ = nullptr;      // 脉压系数    (1 x range_num_)
hipfftComplex* WaveGroupProcessor::d_cfar_coeffs_ = nullptr;    // cfar系数   (1 x range_num_)


WaveGroupProcessor::WaveGroupProcessor(int waveNum, int pulseNum, int rangeNum)
    : wave_num_(waveNum),
      pulse_num_(pulseNum),
      range_num_(rangeNum) {
    allocateDeviceMemory();
    setupFFTPlans();
}

WaveGroupProcessor::~WaveGroupProcessor() {
    cleanup();
    freeDeviceMemory();
    checkCufftErrors(hipfftDestroy(row_plan_));
    checkCufftErrors(hipfftDestroy(col_plan_));
    // checkCufftErrors(hipfftDestroy(pc_plan_));
}

void WaveGroupProcessor::setupFFTPlans() {
    // 创建流
    checkCudaErrors(hipStreamCreate(&stream_));

    // 行FFT (批量处理)

    checkCufftErrors(hipfftPlan1d(&row_plan_, range_num_, HIPFFT_C2C, wave_num_ * pulse_num_));
    checkCufftErrors(hipfftSetStream(row_plan_, stream_));

    // 列FFT (多行处理)
    int rank = 1;
    int n[] = {pulse_num_};
    int inembed[] = {range_num_};
    checkCufftErrors(hipfftPlanMany(&col_plan_, rank, n,
                  inembed, range_num_, 1,
                  inembed, range_num_, 1,
                  HIPFFT_C2C, range_num_));
    checkCufftErrors(hipfftSetStream(col_plan_, stream_));
}

void WaveGroupProcessor::allocateDeviceMemory() {
    const size_t total_size = wave_num_ * pulse_num_ * range_num_;
    currentAddrOffset = 0;
    checkCudaErrors(hipMalloc(&d_unpack_data_, THREADS_MEM_SIZE));
    checkCudaErrors(hipMalloc(&d_headPositions_, sizeof(int) * pulse_num_ * 1.1));
    checkCudaErrors(hipMalloc(&d_data_, sizeof(hipfftComplex) * total_size));
    checkCudaErrors(hipMalloc(&d_cfar_res_, sizeof(hipfftComplex) * total_size));
    checkCudaErrors(hipMalloc(&d_max_results_, sizeof(float) * wave_num_ * range_num_));
    checkCudaErrors(hipMalloc(&d_speed_channels_, sizeof(int) * wave_num_ * range_num_));
}

void WaveGroupProcessor::freeDeviceMemory() {
    checkCudaErrors(hipFree(d_unpack_data_));
    checkCudaErrors(hipFree(d_headPositions_));
    checkCudaErrors(hipFree(d_data_));
    checkCudaErrors(hipFree(d_cfar_res_));
    checkCudaErrors(hipFree(d_max_results_));
    checkCudaErrors(hipFree(d_speed_channels_));
}

void WaveGroupProcessor::cleanup() {
    checkCudaErrors(hipFree(d_pc_coeffs_));
    checkCudaErrors(hipFree(d_cfar_coeffs_));
    checkCufftErrors(hipfftDestroy(pc_plan_));
}

int WaveGroupProcessor::copyRawData(const uint8_t* h_raw_data, size_t data_size)  {

    if ((currentAddrOffset + data_size) <= THREADS_MEM_SIZE) {
        checkCudaErrors(hipMemcpyAsync(d_unpack_data_ + currentAddrOffset,
                                 h_raw_data,
                                 data_size,
                                 hipMemcpyHostToDevice,
                                 stream_));
        currentAddrOffset += data_size;
        return 0;
    }

    currentAddrOffset = 0;
    return -1;
}

void WaveGroupProcessor::getPackegeHeader(uint8_t* h_raw_data, size_t data_size) {
    checkCudaErrors(hipMemcpyAsync(h_raw_data, d_unpack_data_, data_size, hipMemcpyDeviceToHost, stream_));
}

void WaveGroupProcessor::getCoef(std::vector<hipfftComplex>& pcCoef, std::vector<hipfftComplex>& cfarCoef) {

    checkCufftErrors(hipfftPlan1d(&pc_plan_, NFFT, HIPFFT_C2C, 1));

    checkCudaErrors(hipMalloc(&d_pc_coeffs_, NFFT * sizeof(hipfftComplex)));
    checkCudaErrors(hipMalloc(&d_cfar_coeffs_, NFFT * sizeof(hipfftComplex)));
    checkCudaErrors(hipMemcpy(d_pc_coeffs_, pcCoef.data(), NFFT * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_cfar_coeffs_, cfarCoef.data(), NFFT * sizeof(hipfftComplex), hipMemcpyHostToDevice));

    checkCufftErrors(hipfftExecC2C(pc_plan_, d_pc_coeffs_, d_pc_coeffs_, HIPFFT_FORWARD));
    checkCufftErrors(hipfftExecC2C(pc_plan_, d_cfar_coeffs_, d_cfar_coeffs_, HIPFFT_FORWARD));

}

void WaveGroupProcessor::getResult(float* h_max_results_, int* h_speed_channels_) {
    // 选大结果拷贝回内存
    checkCudaErrors(hipMemcpyAsync(h_max_results_, d_max_results_, sizeof(float) * WAVE_NUM * NFFT,
                            hipMemcpyDeviceToHost,
                            stream_));

    // 速度通道拷贝回内存
    checkCudaErrors(hipMemcpyAsync(h_speed_channels_, d_speed_channels_, sizeof(int) * WAVE_NUM * NFFT,
                                    hipMemcpyDeviceToHost,
                                    stream_));
}

void WaveGroupProcessor::unpackData(const int* headPositions) {
    currentAddrOffset = 0;

    checkCudaErrors(hipMemcpyAsync(d_headPositions_, headPositions, pulse_num_ * sizeof(int),
                                hipMemcpyHostToDevice,
                                stream_));

    dim3 gridDim1(wave_num_, (range_num_ + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE, pulse_num_);

    unpackKernel3D<<<gridDim1, CUDA_BLOCK_SIZE, 0, stream_>>>(
        d_unpack_data_, d_data_, d_headPositions_, PULSE_NUM, RANGE_NUM);


}

void WaveGroupProcessor::streamSynchronize() {
    hipStreamSynchronize(stream_);
}

void WaveGroupProcessor::processPulseCompression(int numSamples) {
    int size = wave_num_ * pulse_num_ * range_num_;
    int blockSize = CUDA_BLOCK_SIZE;
    int gridSize = (size + blockSize - 1) / blockSize;

    // fft
    checkCufftErrors(hipfftExecC2C(row_plan_, d_data_, d_data_, HIPFFT_FORWARD));
    // .*
    rowWiseMulKernel<<<gridSize, blockSize, 0, stream_>>>(d_data_, d_pc_coeffs_, wave_num_ * pulse_num_, range_num_);
    // ifft
    checkCufftErrors(hipfftExecC2C(row_plan_, d_data_, d_data_, HIPFFT_BACKWARD));

    this->streamSynchronize();
    writeComplexToFile(d_pc_coeffs_, 1, range_num_, "pccoef.txt");
    writeComplexToFile(d_data_, pulse_num_, range_num_, "2.txt");

    // 设置线程块和网格大小
    int nrows = wave_num_ * pulse_num_;
    int blocksPerGrid = (nrows + blockSize - 1) / blockSize;

    // 启动kernel
    int startIdx = numSamples;
    int endIdx = startIdx + RANGE_NUM - 1;
    moveAndZeroKernel<<<blocksPerGrid, blockSize, 0, stream_>>>(d_data_, nrows, range_num_, startIdx, endIdx);


}

void WaveGroupProcessor::processCoherentIntegration(float scale) {
    // 执行行FFT
    for (int w = 0; w < wave_num_; ++w) {
        hipfftComplex* wavePtr = d_data_ + w * pulse_num_ * range_num_;
        checkCufftErrors(hipfftExecC2C(col_plan_, wavePtr, wavePtr, HIPFFT_FORWARD));
    }

    // 归一化，抵消脉压增益和列fft增益
    int size = wave_num_ * pulse_num_ * range_num_;
    thrust::device_ptr<hipfftComplex> thrust_data(d_data_);
    auto exec_policy = thrust::cuda::par.on(stream_);
    thrust::transform(exec_policy, thrust_data, thrust_data + size, thrust_data, ScaleFunctor(scale));
}

void WaveGroupProcessor::processCFAR() {
    // .^2
    int size = wave_num_ * pulse_num_ * range_num_;
    thrust::device_ptr<hipfftComplex> thrust_data(d_data_);
    auto exec_policy = thrust::cuda::par.on(stream_);
    thrust::transform(exec_policy, thrust_data, thrust_data + size, thrust_data, SquareFunctor());

    // fft
    checkCufftErrors(hipfftExecC2C(row_plan_, d_data_, d_cfar_res_, HIPFFT_FORWARD));
    // .*
    int blockSize = CUDA_BLOCK_SIZE;
    int gridSize = (size + blockSize - 1) / blockSize;
    rowWiseMulKernel<<<gridSize, blockSize, 0, stream_>>>(d_cfar_res_, d_cfar_coeffs_, wave_num_ * pulse_num_, range_num_);
    // ifft
    checkCufftErrors(hipfftExecC2C(row_plan_, d_cfar_res_, d_cfar_res_, HIPFFT_BACKWARD));
    int cfarKernelSize = 2 * numGuardCells + 2 * numRefCells + 1;
    int startIdx = floor((cfarKernelSize - 1) / 2);
    int endIdx = startIdx + RANGE_NUM;

    // 左移抵消卷积扩展
    int nrows = wave_num_ * pulse_num_;
    int blocksPerGrid = (nrows + blockSize - 1) / blockSize;
    moveAndZeroKernel<<<blocksPerGrid, blockSize, 0, stream_>>>(d_cfar_res_, nrows, range_num_, startIdx, endIdx);

    // 根据alpha计算噪底
    double alpha = numRefCells * 2 * (pow(Pfa, -1.0 / (numRefCells * 2)) - 1);
    thrust::device_ptr<hipfftComplex> cfar_data(d_cfar_res_);
    thrust::transform(exec_policy, cfar_data, cfar_data + size, cfar_data, ScaleFunctor(alpha/2.0/numRefCells/pulse_num_));


    // 对比噪底选结果
    cmpKernel<<<gridSize, blockSize, 0, stream_>>>(d_data_, d_cfar_res_, wave_num_ * pulse_num_, range_num_);

    thrust::transform(exec_policy, thrust_data, thrust_data + size, thrust_data, ScaleFunctor(1.0f/normFactor));
    // this->streamSynchronize();
    // writeComplexToFile(d_data_, pulse_num_, range_num_, "2.txt");

}

void WaveGroupProcessor::processMaxSelection() {

    dim3 blockDim_(CUDA_BLOCK_SIZE);
    dim3 gridDim_((range_num_ + blockDim_.x - 1) / blockDim_.x);

    for (int w = 0; w < wave_num_; ++w) {
        auto* cfarPtr = d_data_ + w * pulse_num_ * range_num_;
        float* maxPtr = d_max_results_ + w * range_num_;
        int* speedPtr = d_speed_channels_ + w * range_num_;
        maxKernel<<<gridDim_, blockDim_, 0, stream_>>>(cfarPtr, maxPtr, speedPtr, pulse_num_, range_num_);
    }
}