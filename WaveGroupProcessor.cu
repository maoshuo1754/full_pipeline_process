#include "hip/hip_runtime.h"
#include "WaveGroupProcessor.cuh"
#include "utils.h"
#include "SharedQueue.h"
#include <vector>
#include "kelnels.cuh"
#include "nlohmann/json.hpp"
#include "ThreadPool.h"
#include <matio.h>

WaveGroupProcessor::WaveGroupProcessor(int waveNum, int pulseNum, int rangeNum)
    : wave_num_(waveNum),
      pulse_num_(pulseNum),
      range_num_(rangeNum),
      coef_is_initialized_(false),
      ifft_processor_32(waveNum),
      ifft_processor_8192(azi_densify_crow_num),

      gpu_manager(GpuQueueManager::getInstance())
{
    radar_params_ = new RadarParams();
    allocateDeviceMemory();
    setupFFTPlans();
}

WaveGroupProcessor::~WaveGroupProcessor() {

    freeDeviceMemory();
    checkCufftErrors(hipfftDestroy(row_plan_));
    checkCufftErrors(hipfftDestroy(col_plan_));
    checkCufftErrors(hipfftDestroy(pc_plan_));
}

void WaveGroupProcessor::setupFFTPlans() {
    // 创建流
    checkCudaErrors(hipStreamCreate(&stream_));
    exec_policy_ = thrust::cuda::par.on(stream_);
    // 行FFT (批量处理)
    checkCufftErrors(hipfftPlan1d(&pc_plan_, NFFT, HIPFFT_C2C, 1));
    checkCufftErrors(hipfftPlan1d(&row_plan_, range_num_, HIPFFT_C2C, pulse_num_));
    checkCufftErrors(hipfftSetStream(row_plan_, stream_));

    // 列FFT (多行处理)
    int rank = 1;
    int n[] = {pulse_num_};
    int inembed[] = {range_num_};
    checkCufftErrors(hipfftPlanMany(&col_plan_, rank, n,
                  inembed, range_num_, 1,
                  inembed, range_num_, 1,
                  HIPFFT_C2C, range_num_));
    checkCufftErrors(hipfftSetStream(col_plan_, stream_));
}

void WaveGroupProcessor::allocateDeviceMemory() {
    const size_t total_size = wave_num_ * pulse_num_ * range_num_;
    currentAddrOffset = 0;
    checkCudaErrors(hipMalloc(&d_pc_coeffs_, range_num_ * sizeof(hipfftComplex)));
    checkCudaErrors(hipMalloc(&d_cfar_coeffs_, range_num_ * sizeof(hipfftComplex)));
    checkCudaErrors(hipMalloc(&d_clutterMap_masked_, wave_num_ * pulse_num_ * range_num_));
    checkCudaErrors(hipMemset(d_clutterMap_masked_, 1, wave_num_ * pulse_num_ * range_num_));
    checkCudaErrors(hipMalloc(&d_chnSpeeds, pulse_num_ * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_unpack_data_, THREADS_MEM_SIZE));
    checkCudaErrors(hipMalloc(&d_headPositions_, sizeof(int) * pulse_num_ * 1.1));
    checkCudaErrors(hipMalloc(&d_data_, sizeof(hipfftComplex) * total_size));
    checkCudaErrors(hipMalloc(&d_cfar_res_, sizeof(hipfftComplex) * pulse_num_ * range_num_));
    checkCudaErrors(hipMemset(d_cfar_res_, 0, sizeof(hipfftComplex) * pulse_num_ * range_num_));
    checkCudaErrors(hipMalloc(&d_max_results_, sizeof(float) * wave_num_ * range_num_));
    checkCudaErrors(hipMemset(d_max_results_, 0, sizeof(float) * wave_num_ * range_num_));
    checkCudaErrors(hipMalloc(&d_speed_channels_, sizeof(int) * wave_num_ * range_num_));
    checkCudaErrors(hipMemset(d_speed_channels_, 0, sizeof(int) * wave_num_ * range_num_));
    checkCudaErrors(hipMalloc(&d_detect_rows_, sizeof(int) * pulse_num_));

    // 锁定页内存
    if (hipHostAlloc(&h_data_after_Integration, total_size * sizeof(hipfftComplex), hipHostMallocDefault)!= hipSuccess) {
        std::cerr << "Device memory allocation failed" << std::endl;
        hipHostFree(h_data_after_Integration);
    }

    h_azi_densify_buffer = static_cast<Ipp32fc*>(ippMalloc(azi_densify_crow_num * sizeof(Ipp32fc)));
    h_azi_densify_abs_buffer = static_cast<Ipp32f*>(ippMalloc(azi_densify_crow_num * sizeof(Ipp32f)));
    h_data_after_Integration = new hipfftComplex[total_size];
    thrust_cfar_ = thrust::device_ptr<hipfftComplex>(d_cfar_res_);
}

void WaveGroupProcessor::freeDeviceMemory() {
    checkCudaErrors(hipFree(d_pc_coeffs_));
    checkCudaErrors(hipFree(d_cfar_coeffs_));
    checkCudaErrors(hipFree(d_clutterMap_masked_));
    checkCudaErrors(hipFree(d_chnSpeeds));
    checkCufftErrors(hipfftDestroy(pc_plan_));
    checkCudaErrors(hipFree(d_unpack_data_));
    checkCudaErrors(hipFree(d_headPositions_));
    checkCudaErrors(hipFree(d_data_));
    checkCudaErrors(hipFree(d_cfar_res_));
    checkCudaErrors(hipFree(d_max_results_));
    checkCudaErrors(hipFree(d_speed_channels_));
    checkCudaErrors(hipFree(d_detect_rows_));
    hipHostFree(h_data_after_Integration);

    ippFree(h_azi_densify_buffer);
    ippFree(h_azi_densify_abs_buffer);
}




int WaveGroupProcessor::copyRawData(const uint8_t* h_raw_data, size_t data_size)  {

    if ((currentAddrOffset + data_size) <= THREADS_MEM_SIZE) {
        checkCudaErrors(hipMemcpyAsync(d_unpack_data_ + currentAddrOffset,
                                 h_raw_data,
                                 data_size,
                                 hipMemcpyHostToDevice,
                                 stream_));
        currentAddrOffset += data_size;
        return 0;
    }

    currentAddrOffset = 0;
    return -1;
}

void WaveGroupProcessor::getPackegeHeader(uint8_t* h_raw_data, size_t data_size) {
    checkCudaErrors(hipMemcpyAsync(h_raw_data, d_unpack_data_, data_size, hipMemcpyDeviceToHost, stream_));
}

hipfftComplex* WaveGroupProcessor::getData()
{
    return d_data_;
}

RadarParams* WaveGroupProcessor::getParams()
{
    return radar_params_;
}

void WaveGroupProcessor::getCoef() {
    if (coef_is_initialized_) {
        return;
    }
    coef_is_initialized_ = true;
    clutterMap_range_num_ = ceil(clutter_map_range / radar_params_->distance_resolution) + radar_params_->numSamples + range_correct;
    checkCudaErrors(hipMemcpy(d_pc_coeffs_, radar_params_->pcCoef.data(), NFFT * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_cfar_coeffs_, radar_params_->cfarCoef.data(), NFFT * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_chnSpeeds, radar_params_->chnSpeeds.data(), pulse_num_ * sizeof(int), hipMemcpyHostToDevice));
    detect_rows_num_ = radar_params_->detect_rows.size();
    checkCudaErrors(hipMemcpy(d_detect_rows_, radar_params_->detect_rows.data(), detect_rows_num_ * sizeof(int), hipMemcpyHostToDevice));

    checkCufftErrors(hipfftExecC2C(pc_plan_, d_pc_coeffs_, d_pc_coeffs_, HIPFFT_FORWARD));
    checkCufftErrors(hipfftExecC2C(pc_plan_, d_cfar_coeffs_, d_cfar_coeffs_, HIPFFT_FORWARD));

}

void WaveGroupProcessor::getResult() {
    // 选大结果拷贝回内存
    checkCudaErrors(hipMemcpyAsync(radar_params_->h_max_results_, d_max_results_, sizeof(float) * WAVE_NUM * NFFT,
                            hipMemcpyDeviceToHost,
                            stream_));

    // 速度通道拷贝回内存
    checkCudaErrors(hipMemcpyAsync(radar_params_->h_speed_channels_, d_speed_channels_, sizeof(int) * WAVE_NUM * NFFT,
                                    hipMemcpyDeviceToHost,
                                    stream_));
}

void WaveGroupProcessor::resetAddr()
{
    currentAddrOffset = 0;
}

void WaveGroupProcessor::unpackData(const int* headPositions) {
    checkCudaErrors(hipMemcpyAsync(d_headPositions_, headPositions, pulse_num_ * sizeof(int),
                                hipMemcpyHostToDevice,
                                stream_));

    dim3 gridDim1(wave_num_, (range_num_ + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE, pulse_num_);

    checkCudaErrors(hipMemsetAsync(d_data_, 0, wave_num_ * pulse_num_ * range_num_ * sizeof(hipfftComplex), stream_));
    unpackKernel3D<<<gridDim1, CUDA_BLOCK_SIZE, 0, stream_>>>(
        d_unpack_data_, d_data_, d_headPositions_, PULSE_NUM, RANGE_NUM);

}

void WaveGroupProcessor::streamSynchronize() {
    hipStreamSynchronize(stream_);
}

void WaveGroupProcessor::fullPipelineProcess()
{

    for (cur_wave_ = 0; cur_wave_ < wave_num_; cur_wave_++)
    {

        this->processPulseCompression();

        if (MTI_enable)
        {
            this->processMTI();
        }
        this->processCoherentIntegration(radar_params_->scale);

        if (clutter_map_enable)
        {
            this->processClutterMap();
        }
        this->processCFAR();
        this->processMaxSelection();
    }
}

void WaveGroupProcessor::processPulseCompression() {
    int size = pulse_num_ * range_num_;
    int blockSize = CUDA_BLOCK_SIZE;
    int gridSize = (size + blockSize - 1) / blockSize;

    size_t offset = cur_wave_ * pulse_num_ * range_num_;
    auto* data = d_data_ + offset;

    // fft
    checkCufftErrors(hipfftExecC2C(row_plan_, data, data, HIPFFT_FORWARD));
    // .*
    rowWiseMulKernel<<<gridSize, blockSize, 0, stream_>>>(data, d_pc_coeffs_, pulse_num_, range_num_);
    // ifft
    checkCufftErrors(hipfftExecC2C(row_plan_, data, data, HIPFFT_BACKWARD));
}

void WaveGroupProcessor::processMTI()
{
    dim3 blockDim_(CUDA_BLOCK_SIZE);
    dim3 gridDim_((range_num_ + blockDim_.x - 1) / blockDim_.x);

    auto* waveDataPtr = d_data_ + cur_wave_ * pulse_num_ * range_num_;
    if (MTI_pulse_num == 2)
    {
        MTIkernel2<<<gridDim_, blockDim_, 0, stream_>>>(waveDataPtr, pulse_num_, range_num_);
    }
    else
    {
        MTIkernel3<<<gridDim_, blockDim_, 0, stream_>>>(waveDataPtr, pulse_num_, range_num_);
    }
}

void WaveGroupProcessor::processCoherentIntegration(float scale) {

    size_t offset = cur_wave_ * pulse_num_ * range_num_;
    hipfftComplex* wavePtr = d_data_ + offset;

    checkCufftErrors(hipfftExecC2C(col_plan_, wavePtr, wavePtr, HIPFFT_FORWARD));

    thrust_data_ = thrust::device_ptr<hipfftComplex>(wavePtr);
    // 抵消脉压增益，同时除以range_num_是ifft之后必须除以ifft才能和matlab结果一样
    int size = pulse_num_ * range_num_;
    thrust::transform(exec_policy_, thrust_data_, thrust_data_ + size, thrust_data_, ScaleFunctor(scale / range_num_ / normFactor));

    dim3 blockDim_(CUDA_BLOCK_SIZE);
    dim3 gridDim_((range_num_ + blockDim_.x - 1) / blockDim_.x);

    // 做列 fftshift
    fftshift_columns_inplace_kernel<<<gridDim_, blockDim_, 0, stream_>>>(wavePtr, pulse_num_, range_num_);

    // 拷贝相参积累后的数据到内存，做后续处理
    hipfftComplex* hostPtr = h_data_after_Integration + offset;
    // checkCudaErrors(hipMemcpyAsync(h_data_after_Integration+offset, wavePtr, sizeof(hipfftComplex) * pulse_num_ * range_num_, hipMemcpyDeviceToHost, stream_));

    int range = round(azi_densify_range_end / radar_params_->distance_resolution) + range_correct + radar_params_->numSamples - 1;
    for (int row = radar_params_->detect_rows[0]; row < radar_params_->detect_rows.back(); row++) {
        checkCudaErrors(hipMemcpyAsync(hostPtr + row * range_num_, wavePtr + row * range_num_, sizeof(hipfftComplex) * range, hipMemcpyDeviceToHost, stream_));
    }

}


void WaveGroupProcessor::processClutterMap()
{
    gpu_manager.processClutterMap(d_data_, d_clutterMap_masked_, cur_wave_, clutterMap_range_num_);
}


void WaveGroupProcessor::processCFAR() {
    size_t offset = cur_wave_ * pulse_num_ * range_num_;
    auto* wavePtr = d_data_ + offset;
    // .^2
    int size = pulse_num_ * range_num_;

    thrust_data_ = thrust::device_ptr<hipfftComplex>(wavePtr);


    thrust::transform(exec_policy_, thrust_data_, thrust_data_ + size, thrust_data_, SquareFunctor());

    // fft
    checkCufftErrors(hipfftExecC2C(row_plan_, wavePtr, d_cfar_res_, HIPFFT_FORWARD));
    // .*
    int blockSize = CUDA_BLOCK_SIZE;
    int gridSize = (size + blockSize - 1) / blockSize;
    rowWiseMulKernel<<<gridSize, blockSize, 0, stream_>>>(d_cfar_res_, d_cfar_coeffs_, pulse_num_, range_num_);

    // ifft
    checkCufftErrors(hipfftExecC2C(row_plan_, d_cfar_res_, d_cfar_res_, HIPFFT_BACKWARD));

    thrust::transform(exec_policy_, thrust_cfar_, thrust_cfar_ + size, thrust_cfar_, ScaleFunctor(1.0 / range_num_ ));

    int cfarKernelSize = 2 * numGuardCells + 2 * numRefCells + 1;

    // 用于抵消频域卷积的偏移量
    int shift_offset = floor((cfarKernelSize - 1) / 2);

    // 根据alpha计算噪底
    double alpha = numRefCells * 2 * (pow(Pfa_cfar, -1.0 / (numRefCells * 2)) - 1);
    thrust::transform(exec_policy_, thrust_cfar_, thrust_cfar_ + size, thrust_cfar_, ScaleFunctor(alpha/2.0/numRefCells));

    double* d_rasterize_thresholds_wave = gpu_manager.wave_thresholds(cur_wave_);

    // 对比噪底选结果，(结果开根号)
    cmpKernel<<<gridSize, blockSize, 0, stream_>>>(
        wavePtr,                        // 原始数据
        d_cfar_res_,                    // cfar门限
        d_clutterMap_masked_ + offset,  // 杂波图结果
        pulse_num_,                     // 需要计算的脉冲数
        range_num_,                     // 需要计算的距离单元数
        shift_offset,                   // 平移，抵消频域滤波偏移量
        cfar_enable,                    // cfar控制参数，0代表不做cfar
        d_rasterize_thresholds_wave     // 栅格化门限控制
    );
}

void WaveGroupProcessor::cfar(int numSamples)  {
    double alpha = (numRefCells * 2 * (pow(Pfa_cfar, -1.0 / (numRefCells * 2)) - 1));
    size_t offset = cur_wave_ * pulse_num_ * range_num_;
    // .^2
    int size = pulse_num_ * range_num_;

    auto* wavePtr = d_data_ + offset;
    thrust_data_ = thrust::device_ptr<hipfftComplex>(wavePtr);
    thrust::transform(exec_policy_, thrust_data_, thrust_data_ + size, thrust_data_, SquareFunctor());

    // Configure the CUDA kernel launch parameters
    int colsPerThread = CFAR_LENGTH; // 每个线程处理的列数
    int threadsPerBlock = range_num_ / colsPerThread; // 每个线程块中的线程数
    int blocksPerRow = (range_num_ + colsPerThread - 1) / colsPerThread / threadsPerBlock; // 每行的线程块数
    dim3 blockDim(threadsPerBlock, 1); // 线程块大小：1 行 x 32 列
    int nrows = pulse_num_;
    dim3 gridDim(blocksPerRow, nrows); // 网格大小：每行 block 数 x 总行数

    cfarKernel<<<gridDim, blockDim, 0, stream_>>>(d_data_+offset, d_cfar_res_, nrows, range_num_, alpha, numGuardCells,
                                                  numRefCells, numSamples-1, numSamples+RANGE_NUM-range_correct);
}


void WaveGroupProcessor::processMaxSelection() {
    size_t offset = cur_wave_ * pulse_num_ * range_num_;
    size_t offset2 = cur_wave_ * range_num_;
    // 使用2D block和grid
    dim3 blockDim_(16, 16);  // 可以根据需要调整block大小
    dim3 gridDim_(
        (range_num_ + blockDim_.x - 1) / blockDim_.x,
        1
    );

    if (!rasterize_manage_enable) {
        maxKernel2D<<<gridDim_, blockDim_, 0, stream_>>>(
            d_data_ + offset,           // 输入数据
            d_max_results_ + offset2,    // 最大值输出
            d_speed_channels_ + offset2, // 通道索引输出
            d_chnSpeeds,
            d_detect_rows_,    // 通道范围（row 索引数组）
            detect_rows_num_,  // 通道数量
            pulse_num_,        // 总行数
            range_num_,        // 总列数
            1                  // 总波束数
        );
    }
    else {
        double* d_rasterize_min_speed = gpu_manager.wave_min_speed(cur_wave_);
        double* d_rasterize_max_speed = gpu_manager.wave_max_speed(cur_wave_);

        maxKernel_rasterize<<<gridDim_, blockDim_, 0, stream_>>>(
            d_data_ + offset,           // 输入数据
            d_max_results_ + offset2,    // 最大值输出
            d_speed_channels_ + offset2, // 通道索引输出
            d_chnSpeeds,
            d_rasterize_min_speed,  // 栅格化速度最小索引
            d_rasterize_max_speed,  // 栅格化速度最大索引
            pulse_num_,        // 总行数
            range_num_,        // 总列数
            1                  // 总波束数
        );
    }

}

// 方位加密
void WaveGroupProcessor::processAziDensify() {
    int idx_offset = range_correct + radar_params_->numSamples - 1;
    int start_idx = round(azi_densify_range_start / radar_params_->distance_resolution) + idx_offset;
    int end_idx   = round(azi_densify_range_end / radar_params_->distance_resolution) + idx_offset;

    // 先初始化
    ippsSet_32f(azi_densify_invalid_num, radar_params_->h_azi_densify_results_, WAVE_NUM * NFFT);

    // double max_amp = 0;
    // double est_azi, est_range, est_doppler;
    // int wave_max_idx;

    for (int w = azi_densify_wave_start; w < azi_densify_wave_end; ++w) {
        size_t offset = w * range_num_;
        float* maxresPtr = radar_params_->h_max_results_ + offset;
        int* speedsPtr = radar_params_->h_speed_channels_ + offset;

        Ipp32f maxAmp;
        int maxIdx;
        for (int idx = start_idx; idx <= end_idx; idx++) {
            if (maxresPtr[idx] != 0.0) {
                double targetAziEst = 0;
                double AmpSum = 0;
                memset(h_azi_densify_buffer, 0, azi_densify_crow_num * sizeof(Ipp32fc));
                int doppler_channel = radar_params_->speedsMap[speedsPtr[idx]];
                // Ipp32fc* tmp = new Ipp32fc[wave_num_];
                for (int i = 0; i < wave_num_; i++) {
                    // 自动拷贝到fftshift之后的位置 + (i + wave_num_ / 2) % wave_num_
                    *(h_azi_densify_buffer + (i + wave_num_ / 2) % wave_num_) = *reinterpret_cast<Ipp32fc*>(h_data_after_Integration + i * pulse_num_ * range_num_ + doppler_channel * range_num_ + idx);
                }

                ifft_processor_32.perform_ifft_inplace(h_azi_densify_buffer);  // 32点ifft

                ifft_processor_8192.perform_ifft_inplace(h_azi_densify_buffer); // 8192点ifft

                ifft_processor_8192.fftshift(h_azi_densify_buffer);  // 8192点fftshift

                // save_ipp32fc_to_txt(h_azi_densify_buffer, azi_densify_crow_num, "data1.txt");

                ippsMagnitude_32fc(h_azi_densify_buffer, h_azi_densify_abs_buffer, azi_densify_crow_num);  // 求模

                ippsMaxIndx_32f(h_azi_densify_abs_buffer, azi_densify_crow_num, &maxAmp, &maxIdx);      // 选最大值
                // cout << "maxIdx：" << maxIdx << endl;
                int startIdx = max(maxIdx - azi_densify_EstSample_num, 0);
                int endIdx = min(maxIdx + azi_densify_EstSample_num, azi_densify_crow_num);
                for (int i = startIdx; i < endIdx; i++) {
                    float tmp = h_azi_densify_abs_buffer[i];
                    targetAziEst += radar_params_->h_azi_theta[i] * tmp;
                    AmpSum += tmp;
                }
                radar_params_->h_azi_densify_results_[w * range_num_ + idx] = targetAziEst / AmpSum + 249.0;

                // float wave_azi = getAzi(w, radar_params_->lambda);
                // float AziEst = targetAziEst / AmpSum + 249.0;

                // if (maxresPtr[idx] > max_amp) {
                //     est_doppler = doppler_channel;
                //     wave_max_idx = w;
                //     max_amp = maxresPtr[idx];
                //     est_azi = AziEst;
                //     est_range = (idx - idx_offset);
                // }
                // cout << "wave_num:" << w << endl;
                // cout << "range:" << radar_params_->distance_resolution * (idx - idx_offset) << endl;
                // cout << "originAzi:" << wave_azi << " EstAzi:" << AziEst << " Diff:" << wave_azi - AziEst << endl;
            }
        }
    }
    // cout << endl;
    // cout << "wave: " << wave_max_idx << " est_range: " << est_range << " doppler: " << est_doppler << " azi: " << est_azi << endl;
}


void WaveGroupProcessor::getRadarParams() {
    checkCudaErrors(hipMemcpyAsync(radar_params_->rawMessage, d_unpack_data_, DATA_OFFSET, hipMemcpyDeviceToHost, stream_));

    if (!radar_params_->isInit) {
        radar_params_->isInit = true;
        auto* packageArr = (uint32_t *)(radar_params_->rawMessage);

        auto freqPoint = packageArr[11] & 0x000000ff;
        radar_params_->lambda = c_speed / ((freqPoint * 10 + initCarryFreq) * 1e6);
        radar_params_->pulseWidth = (packageArr[13] & 0xfffff) / Fs_system; //5e-6
        radar_params_->PRT = packageArr[14] / Fs_system;  //125e-6
        auto fLFMStartWord = packageArr[16];
        radar_params_->bandWidth = (Fs_system - fLFMStartWord / pow(2.0f, 32) * Fs_system) * 2.0;
        radar_params_->distance_resolution = c_speed / Fs / 2;

        double fs = 1.0 / radar_params_->PRT;
        double f_step = fs / PULSE_NUM;
        radar_params_->chnSpeeds.clear();

        for(int i = 0; i < PULSE_NUM; ++i) {
            double f = -fs/2.0 + (f_step * i);
            double v = f * radar_params_->lambda / 2.0;
            int v_int = static_cast<int>(std::round(v * 100));
            radar_params_->chnSpeeds.push_back(v_int);
            radar_params_->speedsMap[v_int] = i;
        }

        radar_params_->detect_rows.clear();
        radar_params_->numSamples = round(radar_params_->pulseWidth * Fs);
        for (int row = 0; row < PULSE_NUM; ++row) {
            int speed = std::abs(radar_params_->chnSpeeds[row]);
            if (speed >= v1 && speed <= v2) {
                radar_params_->detect_rows.push_back(row);
            }
        }
        radar_params_->scale = 1.0f / sqrt(radar_params_->bandWidth * radar_params_->pulseWidth) / PULSE_NUM;
        radar_params_->getCoef();
        this->getCoef();
    }
}



void WaveGroupProcessor::saveToDebugFile(int frame, ofstream& debugFile)
{
    if (!debug_mode || frame < start_frame || frame >= end_frame)
    {
        return;
    }

    static bool firstCall = true;  // 静态变量，标记是否为第一次调用
    // 静态成员，用于排序控制
    static std::mutex saveMutex;        // 用于保护 save_cv
    static std::mutex fileMutex;        // 全局互斥锁，用于保护文件写入
    static std::condition_variable save_cv;
    static std::set<int> readyFrames;   // 已准备好保存的 frame
    static int nextToSave = start_frame;         // 下一个待保存的 frame

    // 排序控制逻辑
    {
        std::unique_lock<std::mutex> lock(saveMutex);
        readyFrames.insert(frame);  // 标记当前 frame 已准备好
        // 等待直到当前 frame 是下一个要保存的
        save_cv.wait(lock, [&]{ return frame == nextToSave; });

        // 当前 frame 是 nextToSave，移除并更新 nextToSave
        readyFrames.erase(frame);
        lock.unlock();  // 在文件操作前释放锁
    }

    std::lock_guard<std::mutex> lock(fileMutex);  // 加锁，确保线程安全

    if (firstCall)
    {
        // 定义需要保存的参数
        int pulseNum = PULSE_NUM;  // 假设PULSE_NUM已定义
        int nfft = NFFT;           // 假设NFFT已定义

        // 写入double类型的参数
        debugFile.write(reinterpret_cast<char*>(&radar_params_->bandWidth), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&radar_params_->pulseWidth), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&Fs), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&radar_params_->lambda), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&radar_params_->PRT), sizeof(double));

        // 写入start_wave和end_wave
        debugFile.write(reinterpret_cast<char*>(&start_wave), sizeof(int));
        debugFile.write(reinterpret_cast<char*>(&end_wave), sizeof(int));

        // 写入矩阵大小
        debugFile.write(reinterpret_cast<char*>(&pulseNum), sizeof(int));
        debugFile.write(reinterpret_cast<char*>(&nfft), sizeof(int));

        // 计算并保存32个波束的方位
        std::vector<double> azi(32);
        for (int ii = 0; ii < 32; ++ii)
        {
            int nAzmCode = (azi_table[ii] & 0xffff);
            if (nAzmCode > 32768)
                nAzmCode -= 65536;
            double rAzm = 249.0633 + asin((nAzmCode * radar_params_->lambda) / (65536 * d)) / 3.1415926 * 180.0;
            if (rAzm < 0)
                rAzm += 360.0;
            azi[ii] = rAzm;
        }
        debugFile.write(reinterpret_cast<char*>(azi.data()), 32 * sizeof(double));

        firstCall = false;  // 标记首次写入已完成
    }

    // 以下是原有逻辑，保存当前帧的时间和数据
    int oneWaveSize = PULSE_NUM * NFFT;
    int waveNum = end_wave - start_wave;

    auto* startAddr = d_data_ + start_wave * oneWaveSize;
    size_t size = waveNum * oneWaveSize;

    auto* h_data = new hipfftComplex[size];  // 在主机上分配内存

    // 从显存复制数据到主机内存
    hipMemcpy(h_data, startAddr, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // 写入时间和数据
    auto rawMsg = reinterpret_cast<uint32_t*>(radar_params_->rawMessage);
    auto time = rawMsg[6] / 10 + 8 * 60 * 60 * 1000;  // FPGA时间，0.1ms转为1ms并加8小时

    debugFile.write(reinterpret_cast<char*>(&time), 4);  // 写入时间（4字节）
    debugFile.write(reinterpret_cast<char*>(h_data), size * sizeof(hipfftComplex));  // 写入数据

    delete[] h_data;  // 释放主机内存

    // 更新 nextToSave 并通知其他线程
    {
        std::lock_guard<std::mutex> lock_2(saveMutex);
        nextToSave++;  // 移到下一个 frame
        save_cv.notify_all();  // 通知等待的线程
    }

}


void WaveGroupProcessor::saveToDebugFile_new(int frame, std::string debug_folder_path)
{
    if (!debug_mode || frame < start_frame)
    {
        return;
    }
    static ofstream debugFile;
    if (frame >= end_frame) {
        debugFile.close();
    }

    static bool firstCall = true;  // 静态变量，标记是否为第一次调用
    string message_string = debug_folder_path + "/message.bin";


    if (firstCall)
    {
        firstCall = false; // 标记首次写入已完成
        debugFile.open(message_string, std::ios::binary);
        int pulseNum = PULSE_NUM;
        int nfft = NFFT;

        // 写入double类型的参数
        debugFile.write(reinterpret_cast<char*>(&radar_params_->bandWidth), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&radar_params_->pulseWidth), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&Fs), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&radar_params_->lambda), sizeof(double));
        debugFile.write(reinterpret_cast<char*>(&radar_params_->PRT), sizeof(double));

        // 写入start_frame和end_frame
        debugFile.write(reinterpret_cast<char*>(&start_frame), sizeof(int));
        debugFile.write(reinterpret_cast<char*>(&end_frame), sizeof(int));

        // 写入start_wave和end_wave
        debugFile.write(reinterpret_cast<char*>(&start_wave), sizeof(int));
        debugFile.write(reinterpret_cast<char*>(&end_wave), sizeof(int));

        // 写入矩阵大小
        debugFile.write(reinterpret_cast<char*>(&pulseNum), sizeof(int));
        debugFile.write(reinterpret_cast<char*>(&nfft), sizeof(int));

        // 计算并保存32个波束的方位
        std::vector<double> azi(wave_num_);
        for (int ii = 0; ii < wave_num_; ++ii)
        {
            int nAzmCode = (azi_table[ii] & 0xffff);
            if (nAzmCode > 32768)
                nAzmCode -= 65536;
            double rAzm = 249.0633 + asin((nAzmCode * radar_params_->lambda) / (65536 * d)) / 3.1415926 * 180.0;
            if (rAzm < 0)
                rAzm += 360.0;
            azi[wave_num_ - 1 - ii] = rAzm;
        }
        debugFile.write(reinterpret_cast<char*>(azi.data()), 32 * sizeof(double));
    }



    size_t pulse_num = pulse_num_;
    size_t range_num = range_num_;
    size_t wave_num = end_wave - start_wave;


    size_t oneWaveSize = pulse_num * range_num;
    auto* startAddr = d_data_ + start_wave * oneWaveSize;
    size_t copy_size = wave_num * oneWaveSize;

    auto* h_data = new hipfftComplex[copy_size];  // 在主机上分配内存

    // 从显存复制数据到主机内存
    hipMemcpy(h_data, startAddr, copy_size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // 写入时间和数据
    auto rawMsg = reinterpret_cast<uint32_t*>(radar_params_->rawMessage);
    auto time = rawMsg[6] / 10 + 8 * 60 * 60 * 1000;  // FPGA时间，0.1ms转为1ms并加8小时
    debugFile.write(reinterpret_cast<char*>(&time), 4);  // 写入时间（4字节）

    string cur_mat_path = debug_folder_path + "/frame_" + to_string(frame) + ".mat";
    size_t dims[3] = {pulse_num, range_num, wave_num}; // matlab 读取 的size

    // Allocate arrays for real and imaginary parts
    double* real_data = new double[copy_size];
    double* imag_data = new double[copy_size];

    // Rearrange data into MATLAB column-major order
    size_t idx = 0;
    for (size_t w = 0; w < wave_num; ++w) {
        for (size_t r = 0; r < range_num; ++r) {
            for (size_t p = 0; p < pulse_num; ++p) {
                size_t original_idx = w * oneWaveSize + p * range_num + r; // Original order
                real_data[idx] = static_cast<double>(h_data[original_idx].x);
                imag_data[idx] = static_cast<double>(h_data[original_idx].y);
                idx++;
            }
        }
    }

    // Create .mat file
    mat_t* matfp = Mat_CreateVer(cur_mat_path.c_str(), nullptr, MAT_FT_DEFAULT);

    // Set up complex data structure
    mat_complex_split_t complex_data;
    complex_data.Re = real_data;
    complex_data.Im = imag_data;

    // Create matvar_t structure
    matvar_t* matvar = Mat_VarCreate("data", MAT_C_DOUBLE, MAT_T_DOUBLE, 3, dims, &complex_data, MAT_F_COMPLEX);

    // Write to .mat file
    Mat_VarWrite(matfp, matvar, MAT_COMPRESSION_NONE);

    // Clean up
    Mat_VarFree(matvar);
    Mat_Close(matfp);
    delete[] real_data;
    delete[] imag_data;
    delete[] h_data;  // 释放主机内存
    std::cout << "save " << cur_mat_path << " success" << std::endl;
}