#include "hip/hip_runtime.h"
#include "WaveGroupProcessor.h"
#include "utils.h"
#include "SharedQueue.h"

WaveGroupProcessor::WaveGroupProcessor(int waveNum, int pulseNum, int rangeNum, hipStream_t stream)
    : wave_num_(waveNum),
      pulse_num_(pulseNum),
      range_num_(rangeNum),
      stream_(stream) {
    allocateDeviceMemory();
    setupFFTPlans();

    // 初始化中间结果矩阵
    for (int i = 0; i < wave_num_; ++i) {
        cfars_.emplace_back(pulse_num_, range_num_);
        temps_.emplace_back(pulse_num_, range_num_);
    }
}

WaveGroupProcessor::~WaveGroupProcessor() {
    freeDeviceMemory();
    hipfftDestroy(row_plan_);
    hipfftDestroy(col_plan_);
    hipfftDestroy(pc_plan_);
}

void WaveGroupProcessor::setupFFTPlans() {
    // 脉压FFT (按行)
    hipfftPlan1d(&pc_plan_, range_num_, HIPFFT_C2C, pulse_num_);
    hipfftSetStream(pc_plan_, stream_);

    // 行FFT (批量处理)
    hipfftPlan1d(&row_plan_, range_num_, HIPFFT_C2C, pulse_num_);
    hipfftSetStream(row_plan_, stream_);

    // 列FFT (多行处理)
    int rank = 1;
    int n[] = {pulse_num_};
    int inembed[] = {range_num_};
    hipfftPlanMany(&col_plan_, rank, n,
                  inembed, range_num_, 1,
                  inembed, range_num_, 1,
                  HIPFFT_C2C, range_num_);
    hipfftSetStream(col_plan_, stream_);
}

void WaveGroupProcessor::allocateDeviceMemory() {
    const size_t total_size = wave_num_ * pulse_num_ * range_num_;
    hipMalloc(&d_data_, sizeof(hipfftComplex) * total_size);
    hipMalloc(&d_max_results_, sizeof(hipfftComplex) * wave_num_ * range_num_);
    hipMalloc(&d_speed_channels_, sizeof(int) * wave_num_ * range_num_);
}

void WaveGroupProcessor::freeDeviceMemory() {
    hipFree(d_data_);
    hipFree(d_max_results_);
    hipFree(d_speed_channels_);
}

__global__ void unpackKernel3D(unsigned char* raw, hipfftComplex* output,
                              const int* heads, int waveNum, int pulseNum, int rangeNum) {
    const int waveIdx = blockIdx.z;
    const int pulseIdx = blockIdx.y;
    const int rangeIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (rangeIdx >= rangeNum) return;

    const int headOffset = heads[pulseIdx];
    unsigned char* blockStart = raw + headOffset + DATA_OFFSET;
    const int byteOffset = rangeIdx * waveNum * 4 + waveIdx * 4;
    
    output[(waveIdx * pulseNum + pulseIdx) * rangeNum + rangeIdx].x = 
        __int2half_rn(*(int16_t*)(blockStart + byteOffset + 2));
    output[(waveIdx * pulseNum + pulseIdx) * rangeNum + rangeIdx].y = 
        __int2half_rn(*(int16_t*)(blockStart + byteOffset));
}

void WaveGroupProcessor::unpackData(unsigned char* rawData, const int* headPositions, int numHeads) {
    dim3 block(256);
    dim3 grid((range_num_ + block.x - 1) / block.x, pulse_num_, wave_num_);
    unpackKernel3D<<<grid, block, 0, stream_>>>(
        rawData, d_data_, headPositions, wave_num_, pulse_num_, range_num_);
}

__global__ void batchPulseCompression(hipfftComplex* data, const hipfftComplex* pcCoef,
                                      int waveNum, int pulseNum, int rangeNum) {
    const int waveIdx = blockIdx.z;
    const int pulseIdx = blockIdx.y;
    const int rangeIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (rangeIdx >= rangeNum) return;

    hipfftComplex* waveData = data + waveIdx * pulseNum * rangeNum;
    hipfftComplex val = waveData[pulseIdx * rangeNum + rangeIdx];
    hipfftComplex coef = pcCoef[rangeIdx];
    
    // 频域相乘
    waveData[pulseIdx * rangeNum + rangeIdx] = hipCmulf(val, coef);
}

void WaveGroupProcessor::processPulseCompression(const CudaMatrix& pcCoefMatrix) {
    // 执行行FFT
    for (int w = 0; w < wave_num_; ++w) {
        hipfftComplex* wavePtr = d_data_ + w * pulse_num_ * range_num_;
        hipfftExecC2C(row_plan_, wavePtr, wavePtr, HIPFFT_FORWARD);
    }

    // 频域相乘
    dim3 block(256);
    dim3 grid((range_num_ + block.x - 1) / block.x, pulse_num_, wave_num_);
    batchPulseCompression<<<grid, block, 0, stream_>>>(
        d_data_, pcCoefMatrix.getData(), wave_num_, pulse_num_, range_num_);

    // 执行逆FFT
    for (int w = 0; w < wave_num_; ++w) {
        hipfftComplex* wavePtr = d_data_ + w * pulse_num_ * range_num_;
        hipfftExecC2C(row_plan_, wavePtr, wavePtr, HIPFFT_BACKWARD);
    }
}

// 其他处理函数实现类似，使用三维核函数处理所有波束...