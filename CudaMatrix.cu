#include "hip/hip_runtime.h"
#include "CudaMatrix.h"
#include <iostream>
#include <stdexcept>
#include <cmath>
#include <hip/hip_complex.h>

using namespace std;

// Helper functions to check errors
void CudaMatrix::checkCudaErrors(hipError_t result) {
    if (result != hipSuccess) {
        throw runtime_error(hipGetErrorString(result));
    }
}

void CudaMatrix::checkCublasErrors(hipblasStatus_t result) {
    if (result != HIPBLAS_STATUS_SUCCESS) {
        throw runtime_error("cuBLAS error");
    }
}

void CudaMatrix::checkCufftErrors(hipfftResult result) {
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: ";
        switch (result) {
            case HIPFFT_INVALID_PLAN:
                std::cerr << "HIPFFT_INVALID_PLAN";
                break;
            case HIPFFT_ALLOC_FAILED:
                std::cerr << "HIPFFT_ALLOC_FAILED";
                break;
            case HIPFFT_INVALID_TYPE:
                std::cerr << "HIPFFT_INVALID_TYPE";
                break;
            case HIPFFT_INVALID_VALUE:
                std::cerr << "HIPFFT_INVALID_VALUE";
                break;
            case HIPFFT_INTERNAL_ERROR:
                std::cerr << "HIPFFT_INTERNAL_ERROR";
                break;
            case HIPFFT_EXEC_FAILED:
                std::cerr << "HIPFFT_EXEC_FAILED";
                break;
            case HIPFFT_SETUP_FAILED:
                std::cerr << "HIPFFT_SETUP_FAILED";
                break;
            case HIPFFT_INVALID_SIZE:
                std::cerr << "HIPFFT_INVALID_SIZE";
                break;
            default:
                std::cerr << "Unknown error";
        }
        std::cerr << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Constructor
CudaMatrix::CudaMatrix(int rows, int cols) : nrows(rows), ncols(cols), data(nullptr) {
    allocateMemory();
}

// Constructor for creating a matrix from host data
CudaMatrix::CudaMatrix(const std::vector<std::vector<hipfftComplex>>& hostData)
        : nrows(hostData.size()), ncols(hostData[0].size()), data(nullptr) {
    allocateMemory();
    // Copy data from host to device
    std::vector<hipfftComplex> flattenedData(nrows * ncols);
    for (int i = 0; i < nrows; ++i) {
        std::copy(hostData[i].begin(), hostData[i].end(), flattenedData.begin() + i * ncols);
    }
    checkCudaErrors(hipMemcpy(data, flattenedData.data(), sizeof(hipfftComplex) * nrows * ncols, hipMemcpyHostToDevice));
}

CudaMatrix::CudaMatrix(int rows, int cols, std::vector<hipfftComplex> hostData) : nrows(rows), ncols(cols) {
    if (hostData.size() != nrows * ncols) {
        throw std::runtime_error("Host data size does not match matrix dimensions.");
    }
    checkCudaErrors(hipMalloc(&data, sizeof(hipfftComplex) * nrows * ncols));
    checkCudaErrors(hipMemcpy(data, hostData.data(), sizeof(hipfftComplex) * nrows * ncols, hipMemcpyHostToDevice));
}

CudaMatrix::CudaMatrix(int rows, int cols, hipfftComplex* hostData) : nrows(rows), ncols(cols) {
    checkCudaErrors(hipMalloc(&data, sizeof(hipfftComplex) * nrows * ncols));
    checkCudaErrors(hipMemcpy(data, hostData, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyHostToDevice));
}

// Copy constructor
CudaMatrix::CudaMatrix(const CudaMatrix& other) : nrows(other.nrows), ncols(other.ncols), data(nullptr) {
//    cout <<"copy constructor" << endl;
    allocateMemory();
    checkCudaErrors(hipMemcpy(data, other.data, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyDeviceToDevice));
}

// Move constructor
CudaMatrix::CudaMatrix(CudaMatrix&& other) noexcept : nrows(other.nrows), ncols(other.ncols), data(other.data) {
    other.data = nullptr;
    other.nrows = 0;
    other.ncols = 0;
}


// Destructor
CudaMatrix::~CudaMatrix() {
    deallocateMemory();
//    cout << "Destructor" << endl;
}

// Copy data from host to device
void CudaMatrix::copyFromHost(const vector<hipfftComplex> &hostData) {
    if (hostData.size() != nrows * ncols) {
        throw runtime_error("Host data size does not match matrix size.");
    }
    checkCudaErrors(hipMemcpy(data, hostData.data(), sizeof(hipfftComplex) * nrows * ncols, hipMemcpyHostToDevice));
}

void CudaMatrix::copyFromHost(int rows, int cols, const hipfftComplex* hostData) {
    if (rows != nrows || cols != nrows) {
        deallocateMemory();
        nrows = rows;
        ncols = cols;
        allocateMemory();
    }
    checkCudaErrors(hipMemcpy(data, hostData, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyHostToDevice));
}

// Copy data from device to host
void CudaMatrix::copyToHost(vector<hipfftComplex> &hostData) const {
    hostData.resize(nrows * ncols);
    checkCudaErrors(hipMemcpy(hostData.data(), data, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyDeviceToHost));
}

// Get matrix data as 2D vector
vector<vector<hipfftComplex>> CudaMatrix::to2DVector() const {
    vector<hipfftComplex> flatData;
    copyToHost(flatData);
    vector<vector<hipfftComplex>> result(nrows, vector<hipfftComplex>(ncols));

    for (int i = 0; i < nrows; ++i) {
        for (int j = 0; j < ncols; ++j) {
            result[i][j] = flatData[i * ncols + j];
        }
    }
    return result;
}

// Fill matrix with random values
void CudaMatrix::fillWithRandomValues() {
    vector<hipfftComplex> hostData(nrows * ncols);
    for (int i = 0; i < nrows * ncols; ++i) {
        hostData[i].x = static_cast<float>(rand()) / RAND_MAX;
        hostData[i].y = static_cast<float>(rand()) / RAND_MAX;
    }
    checkCudaErrors(hipMemcpy(data, hostData.data(), nrows * ncols * sizeof(hipfftComplex), hipMemcpyHostToDevice));
}

// Print matrix for debugging
void CudaMatrix::print() const {
    vector<hipfftComplex> hostData;
    copyToHost(hostData);

    cout << "array([" << endl;
    for (int i = 0; i < nrows; ++i) {
        cout << "  [";
        for (int j = 0; j < ncols; ++j) {
            cout << std::fixed << std::setprecision(5) << std::setw(6)
                 << hostData[i * ncols + j].x << " + "
                 << std::fixed << std::setprecision(5) << std::setw(6)
                 << hostData[i * ncols + j].y << "j";
            if (j < ncols - 1) {
                cout << ",  ";
            }
        }
        cout << "]";
        if (i < nrows - 1) {
            cout << ",";
        }
        cout << endl;
    }
    cout << "])" << endl;
}

void CudaMatrix::print(int row) const {
    if (row >= nrows || row < 0) {
        throw runtime_error("Index out of bounds.");
    }

    vector<hipfftComplex> hostData;
    copyToHost(hostData);

    cout << "array of row" << row << "([" << endl;
    int i = row;
    cout << "  [";
    for (int j = 0; j < ncols; ++j) {
        cout << std::fixed << std::setprecision(5) << std::setw(6)
             << hostData[i * ncols + j].x << " + "
             << std::fixed << std::setprecision(5) << std::setw(6)
             << hostData[i * ncols + j].y << "j";
        if (j < ncols - 1) {
            cout << ",  ";
        }
    }
    cout << "]";
    if (i < nrows - 1) {
        cout << ",";
    }
    cout << endl;

    cout << "])" << endl;
}

void CudaMatrix::printLargerThan0() const {
    vector<hipfftComplex> hostData;
    copyToHost(hostData);

    for (int i = 0; i < this->size(); i++) {
        if (hostData[i].x > 0) {
            cout << i << ":" << hostData[i].x << endl;
        }
    }
}

void CudaMatrix::printShape() const {
    cout << "CudaMatrix Shape: (" << nrows << ", " << ncols << ")" << endl;
}

void CudaMatrix::setElement(int x, int y, hipfftComplex value) {
    if (x >= nrows || y >= ncols || x < 0 || y < 0) {
        throw runtime_error("Index out of bounds.");
    }
    hipMemcpy(data + x * ncols + y, &value, sizeof(hipfftComplex), hipMemcpyHostToDevice);
}

// Kernel for element-wise multiplication
__global__ void elementWiseMulKernel(hipfftComplex *d_a, hipfftComplex *d_b, hipfftComplex *d_c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hipfftComplex temp_a = d_a[idx];
        hipfftComplex temp_b = d_b[idx];
        d_c[idx].x = temp_a.x * temp_b.x - temp_a.y * temp_b.y;
        d_c[idx].y = temp_a.x * temp_b.y + temp_a.y * temp_b.x;
    }
}

// Element-wise multiplication method
CudaMatrix CudaMatrix::elementWiseMul(const CudaMatrix &other, bool inplace) const {
    if (nrows != other.nrows || ncols != other.ncols) {
        throw std::runtime_error("Matrix dimensions must match for element-wise multiplication.");
    }

    int size = nrows * ncols;
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    if (inplace) {
        elementWiseMulKernel<<<gridSize, blockSize>>>(data, other.data, data, size);
        hipDeviceSynchronize();
        return {};
    } else {
        CudaMatrix result(nrows, ncols);
        elementWiseMulKernel<<<gridSize, blockSize>>>(data, other.data, result.data, size);
        hipDeviceSynchronize();
        return result;
    }
}


// Kernel for transpose
__global__ void transposeKernel(hipfftComplex* input, hipfftComplex* output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        output[x * rows + y] = input[y * cols + x];
    }
}

CudaMatrix CudaMatrix::T(bool inplace) {
    // Non in-place transpose
    CudaMatrix result(ncols, nrows);

    dim3 blockSize(16, 16);
    dim3 gridSize((ncols + blockSize.x - 1) / blockSize.x, (nrows + blockSize.y - 1) / blockSize.y);
    transposeKernel<<<gridSize, blockSize>>>(data, result.data, nrows, ncols);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    if(inplace){
        *this = std::move(result);
        return {};
    }
    return result;
}

// Kernel for matrix multiplication
__global__ void
matMulKernel(const hipfftComplex *A, const hipfftComplex *B, hipfftComplex *C, int Arows, int Acols, int Bcols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < Arows && col < Bcols) {
        hipfftComplex sum = make_hipComplex(0.0f, 0.0f);
        for (int i = 0; i < Acols; ++i) {
            hipfftComplex a = A[row * Acols + i];
            hipfftComplex b = B[i * Bcols + col];
            sum.x += a.x * b.x - a.y * b.y;
            sum.y += a.x * b.y + a.y * b.x;
        }
        C[row * Bcols + col] = sum;
    }
}

// Matrix multiplication method using custom kernel
CudaMatrix CudaMatrix::operator*(const CudaMatrix &other) const {
    if (ncols != other.nrows) {
        throw runtime_error("Matrix dimensions must match for multiplication.");
    }

    CudaMatrix result(nrows, other.ncols);

    dim3 blockSize(16, 16);
    dim3 gridSize((other.ncols + blockSize.x - 1) / blockSize.x, (nrows + blockSize.y - 1) / blockSize.y);
    matMulKernel<<<gridSize, blockSize>>>(data, other.data, result.data, nrows, ncols, other.ncols);

    checkCudaErrors(hipDeviceSynchronize());

    return result;
}

bool CudaMatrix::operator==(const CudaMatrix& other) const {
    if (nrows != other.nrows || ncols != other.ncols) {
        return false;
    }

    std::vector<hipfftComplex> hostDataThis(nrows * ncols);
    std::vector<hipfftComplex> hostDataOther(nrows * ncols);

    checkCudaErrors(hipMemcpy(hostDataThis.data(), data, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hostDataOther.data(), other.data, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyDeviceToHost));

    const float epsilon = 1e-6;

    for (int i = 0; i < nrows * ncols; ++i) {
        if (std::fabs(hostDataThis[i].x - hostDataOther[i].x) > epsilon ||
            std::fabs(hostDataThis[i].y - hostDataOther[i].y) > epsilon) {
            return false;
        }
    }

    return true;
}

// Copy assignment operator
CudaMatrix& CudaMatrix::operator=(const CudaMatrix& other) {
    if (this != &other) {
        deallocateMemory();
        nrows = other.nrows;
        ncols = other.ncols;
        allocateMemory();
        checkCudaErrors(hipMemcpy(data, other.data, sizeof(hipfftComplex) * nrows * ncols, hipMemcpyDeviceToDevice));
    }
    return *this;
}

// Move assignment operator
CudaMatrix& CudaMatrix::operator=(CudaMatrix&& other) noexcept {
    if (this != &other) {
        deallocateMemory();
        nrows = other.nrows;
        ncols = other.ncols;
        data = other.data;
        other.data = nullptr;
        other.nrows = 0;
        other.ncols = 0;
    }
    return *this;
}

CudaMatrix CudaMatrix::fft(bool inplace) const{
    if (inplace) {
        hipfftHandle plan;
        checkCufftErrors(hipfftPlan1d(&plan, ncols, HIPFFT_C2C, nrows));
        checkCufftErrors(hipfftExecC2C(plan, data, data, HIPFFT_FORWARD));
        checkCufftErrors(hipfftDestroy(plan));
        return {};
    } else {
        CudaMatrix result(nrows, ncols);
        hipfftHandle plan;
        checkCufftErrors(hipfftPlan1d(&plan, ncols, HIPFFT_C2C, nrows));
        checkCufftErrors(hipfftExecC2C(plan, data, result.data, HIPFFT_FORWARD));
        checkCufftErrors(hipfftDestroy(plan));
        return result;
    }
}

CudaMatrix CudaMatrix::fft_by_col(bool inplace){
    hipfftHandle plan;

    // 按列做FFT
    int batch = ncols;
    int inembed[] = {ncols};  // 每行数据的大小
    int onembed[] = {ncols};  // 每行数据的大小
    int istride = ncols;          // 每个元素步幅
    int idist = 1;                // 每行之间的距离
    int ostride = ncols;          // 每个元素步幅
    int odist = 1;                // 每行之间的距离
    int n[] = {nrows};            // 每行数据的FFT大小

    checkCufftErrors(hipfftPlanMany(&plan, 1, n, // Rank and size of the FFT
                           inembed, istride, idist,   // Input data layout
                           onembed, ostride, odist,   // Output data layout
                           HIPFFT_C2C, batch)   // FFT type and number of FFTs
    );
    if (inplace) {
        checkCufftErrors(hipfftExecC2C(plan, data, data, HIPFFT_FORWARD));
        checkCufftErrors(hipfftDestroy(plan));
        return {};
    } else {
        CudaMatrix result(nrows, ncols);
        checkCufftErrors(hipfftExecC2C(plan, data, result.data, HIPFFT_FORWARD));
        checkCufftErrors(hipfftDestroy(plan));
        return result;
    }
}



__global__ void zeroPadAndCopy(hipfftComplex* idata, hipfftComplex* odata, int nrows, int ncols, int nPoints) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < nrows) {
        int rowOffsetSrc = row * ncols;
        int rowOffsetDst = row * nPoints;
        for (int col = 0; col < ncols; ++col) {
            odata[rowOffsetDst + col] = idata[rowOffsetSrc + col];
        }
        for (int col = ncols; col < nPoints; ++col) {
            odata[rowOffsetDst + col].x = 0.0f;
            odata[rowOffsetDst + col].y = 0.0f;
        }
    }
}

CudaMatrix CudaMatrix::fft_N(int nPoints, bool inplace) {
    if (nPoints < ncols) {
        throw std::runtime_error(
                "Number of FFT points must be greater than or equal to the number of columns in the matrix.");
    }

    // Create a result matrix with the same number of rows and the specified number of points
    CudaMatrix result(nrows, nPoints);

    // Set up CUDA kernel configuration
    int threadsPerBlock = 128;
    int blocksPerGrid = (nrows + threadsPerBlock - 1) / threadsPerBlock;

    // Launch CUDA kernel to copy data and zero-pad
    zeroPadAndCopy<<<blocksPerGrid, threadsPerBlock>>>(data, result.data, nrows, ncols, nPoints);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Create a hipfftHandle for the batched FFT plan
    hipfftHandle plan;
    checkCufftErrors(hipfftPlan1d(&plan, nPoints, HIPFFT_C2C, nrows));

    // Execute the FFT plan
    checkCufftErrors(hipfftExecC2C(plan, result.data, result.data, HIPFFT_FORWARD));

    // Clean up
    checkCufftErrors(hipfftDestroy(plan));

    if(inplace){
        *this = std::move(result);
        return {};
    }
    return result;
}


struct ScaleFunctor {
    float scale;

    ScaleFunctor(float s) : scale(s) {}

    __device__ hipfftComplex operator()(hipfftComplex c) const {
        return make_hipComplex(c.x * scale, c.y * scale);
    }
};

CudaMatrix CudaMatrix::ifft(bool inplace) const {
    hipfftHandle plan;
    checkCufftErrors(hipfftPlan1d(&plan, ncols, HIPFFT_C2C, nrows));


    if (inplace) {
        checkCufftErrors(hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD));
        checkCufftErrors(hipfftDestroy(plan));
        // Scale the result by 1/ncols to get the correct IFFT result
        float scale = 1.0f / ncols;
        thrust::device_ptr<hipfftComplex> thrust_data(data);
        thrust::transform(thrust_data, thrust_data + nrows * ncols, thrust_data, ScaleFunctor(scale));
        return {};
    } else {
        CudaMatrix result(nrows, ncols);
        checkCufftErrors(hipfftExecC2C(plan, data, result.data, HIPFFT_BACKWARD));
        checkCufftErrors(hipfftDestroy(plan));
        // Scale the result by 1/ncols to get the correct IFFT result
        float scale = 1.0f / ncols;
        thrust::device_ptr<hipfftComplex> thrust_result(result.data);
        thrust::transform(thrust_result, thrust_result + nrows * ncols, thrust_result, ScaleFunctor(scale));

        return result;
    }
}



// Kernel to extract segments from each row
__global__ void
extractSegmentKernel(hipfftComplex *input, hipfftComplex *output, int nrows, int ncols, int startInd, int rangeNumber) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < nrows) {
        for (int j = 0; j < rangeNumber; ++j) {
            int inputIdx = row * ncols + startInd + j;
            int outputIdx = row * rangeNumber + j;
            output[outputIdx] = input[inputIdx];
        }
    }
}

CudaMatrix CudaMatrix::extractSegment(int startInd, int rangeNumber) const {
    if (startInd < 0 || startInd + rangeNumber > ncols) {
        throw std::runtime_error("Index out of bounds.");
    }

    CudaMatrix result(nrows, rangeNumber);

    // Launch kernel to extract segments
    int blockSize = 256;
    int gridSize = (nrows + blockSize - 1) / blockSize;
    extractSegmentKernel<<<gridSize, blockSize>>>(data, result.data, nrows, ncols, startInd, rangeNumber);

    // Check for errors
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    return result;
}

void CudaMatrix::writeMatTxt(const std::string &filePath) const {
    std::ofstream outfile(filePath);

    if (!outfile.is_open()) {
        std::cerr << "Error opening file " << filePath << " for writing!" << std::endl;
        return;
    }

    outfile << nrows << " " << ncols << std::endl;

        std::vector<hipfftComplex> hostData(nrows * ncols);
    copyToHost(hostData);

    for (int i = 0; i < nrows; ++i) {
        for (int j = 0; j < ncols; ++j) {
            outfile << hostData[i * ncols + j].x << " " << hostData[i * ncols + j].y << endl;
        }
        outfile << std::endl;
    }

    outfile.close();
}

__global__ void cfarKernel(const hipfftComplex* data, hipfftComplex* cfar_signal, int nrows, int ncols, double alpha, int numGuardCells, int numRefCells) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < nrows && col < ncols) {
        int total_training_cells = numGuardCells + numRefCells;

        double noise_level = 0.0;
        int num_ref_cells;

        if (col < total_training_cells) {
            for (int i = col + numGuardCells + 1; i <= col + numGuardCells + numRefCells; ++i) {
                if (i < ncols) {
                    noise_level += data[row * ncols + i].x;
                }
            }
            num_ref_cells = numRefCells;
        } else if (col >= ncols - total_training_cells) {
            for (int i = col - numRefCells - numGuardCells; i < col - numGuardCells; ++i) {
                if (i >= 0) {
                    noise_level += data[row * ncols + i].x;
                }
            }
            num_ref_cells = numRefCells;
        } else {
            for (int i = col - total_training_cells; i < col - total_training_cells + numRefCells; ++i) {
                noise_level += data[row * ncols + i].x;
            }
            for (int i = col + numGuardCells + 1; i <= col + numGuardCells + numRefCells; ++i) {
                noise_level += data[row * ncols + i].x;
            }
            num_ref_cells = numRefCells * 2;
        }

        double threshold = alpha * noise_level / num_ref_cells;

        if (data[row * ncols + col].x > threshold) {
            cfar_signal[row * ncols + col].x = sqrt(data[row * ncols + col].x);
        }
    }
}

CudaMatrix CudaMatrix::cfar(double Pfa, int numGuardCells, int numRefCells) const {
    double alpha = (numRefCells * 2 * (pow(Pfa, -1.0 / (numRefCells * 2)) - 1));

    // Compute the absolute values
    CudaMatrix absData = this->abs(false);

    // Compute the squared absolute values
    absData.elementWiseSquare();

    // Initialize the CFAR result matrix
    CudaMatrix cfar_signal(nrows, ncols);

    // Configure the CUDA kernel launch parameters
    dim3 blockDim(16, 16);
    dim3 gridDim((ncols + blockDim.x - 1) / blockDim.x, (nrows + blockDim.y - 1) / blockDim.y);

    // Launch the CFAR kernel
    cfarKernel<<<gridDim, blockDim>>>(absData.data, cfar_signal.data, nrows, ncols, alpha, numGuardCells, numRefCells);
    hipDeviceSynchronize();
    return cfar_signal;
}

// Helper kernels
__global__ void maxKernelDim1(hipfftComplex *data, hipfftComplex *maxValues, int nrows, int ncols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ncols) {
        hipfftComplex maxVal = data[col];
        for (int row = 1; row < nrows; ++row) {
            hipfftComplex val = data[row * ncols + col];
            if (sqrt(val.x * val.x + val.y * val.y) > sqrt(maxVal.x * maxVal.x + maxVal.y * maxVal.y)) {
                maxVal = val;
            }
        }
        maxValues[col] = maxVal;
    }
}

__global__ void maxKernelDim2(hipfftComplex *data, hipfftComplex *maxValues, int nrows, int ncols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < nrows) {
        hipfftComplex maxVal = data[row * ncols];
        for (int col = 1; col < ncols; ++col) {
            hipfftComplex val = data[row * ncols + col];
            if (sqrt(val.x * val.x + val.y * val.y) > sqrt(maxVal.x * maxVal.x + maxVal.y * maxVal.y)) {
                maxVal = val;
            }
        }
        maxValues[row] = maxVal;
    }
}

CudaMatrix CudaMatrix::max(int dim) {
    assert(dim == 1 || dim == 2); // Ensure valid dimension


    if (dim == 1) {
        auto result = CudaMatrix(1, ncols); // Result will be 1 row, ncols columns
        hipfftComplex *d_maxValues;
        hipMalloc(&d_maxValues, ncols * sizeof(hipfftComplex));

        dim3 blockDim(256);
        dim3 gridDim((ncols + blockDim.x - 1) / blockDim.x);

        maxKernelDim1<<<gridDim, blockDim>>>(data, d_maxValues, nrows, ncols);
        hipDeviceSynchronize();

        hipMemcpy(result.data, d_maxValues, ncols * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
        hipFree(d_maxValues);
        return result;
    } else {
        auto result = CudaMatrix(nrows, 1); // Result will be nrows rows, 1 column
        hipfftComplex *d_maxValues;
        hipMalloc(&d_maxValues, nrows * sizeof(hipfftComplex));

        dim3 blockDim(256);
        dim3 gridDim((nrows + blockDim.x - 1) / blockDim.x);

        maxKernelDim2<<<gridDim, blockDim>>>(data, d_maxValues, nrows, ncols);
        hipDeviceSynchronize();

        hipMemcpy(result.data, d_maxValues, nrows * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
        hipFree(d_maxValues);
        return result;
    }

}

__global__ void elementWiseSquareKernel(hipfftComplex *idata, hipfftComplex *odata, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hipfftComplex value = idata[idx];
        odata[idx].x = value.x * value.x - value.y * value.y;
        odata[idx].y = 2 * value.x * value.y;
    }
}

CudaMatrix CudaMatrix::elementWiseSquare(bool inplace) const {
    int blockSize = 256;
    int size = ncols * nrows;
    int gridSize = (size + blockSize - 1) / blockSize;

    if (inplace) {
        elementWiseSquareKernel<<<gridSize, blockSize>>>(data, data, size);
        hipDeviceSynchronize();
        return {};
    } else {
        CudaMatrix result(this->nrows, this->ncols);
        elementWiseSquareKernel<<<gridSize, blockSize>>>(data, result.data, size);
        hipDeviceSynchronize();
        return result;
    }
}

__global__ void absCufftComplexKernel(hipfftComplex *idata, hipfftComplex *odata, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        odata[idx].x = hipCabsf(idata[idx]);
        odata[idx].y = 0;
    }
}


CudaMatrix CudaMatrix::abs(bool inplace) const {
    int blockSize = 256;
    int size = ncols * nrows;
    int gridSize = (size + blockSize - 1) / blockSize;

    if (inplace) {
        absCufftComplexKernel<<<gridSize, blockSize>>>(data, data, size);
        hipDeviceSynchronize();
        return {};
    } else {
        CudaMatrix result(this->nrows, this->ncols);
        absCufftComplexKernel<<<gridSize, blockSize>>>(data, result.data, size);
        hipDeviceSynchronize();
        return result;
    }
}

// Allocate memory on the device
void CudaMatrix::allocateMemory() {
    if (nrows > 0 && ncols > 0) {
        checkCudaErrors(hipMalloc(&data, sizeof(hipfftComplex) * nrows * ncols));
    }
}

// Deallocate memory on the device
void CudaMatrix::deallocateMemory() {
    if (data) {
        checkCudaErrors(hipFree(data));
        data = nullptr;
    }
}

void CudaMatrix::setSize(int _nrows, int _ncols) {
    if (data != nullptr){
        deallocateMemory();
        data = nullptr;
    }
    nrows = _nrows;
    ncols = _ncols;

    if (nrows > 0 && ncols > 0) {
        checkCudaErrors(hipMalloc(&data, sizeof(hipfftComplex) * nrows * ncols));
    }
}
